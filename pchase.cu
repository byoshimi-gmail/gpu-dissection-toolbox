
#include <hip/hip_runtime.h>
#include <stdint.h>

#define WARP_SIZE 8
#define L1_SIZE 32
#define THREADS_NUM 1

/// The following are a number renditions of the P-Chase algorithm from a number
/// of publications: Dissecting the NVIDIA Volta GPU Architecture via
/// Microbenchmarking: https://arxiv.org/abs/1804.06826 Dissecting GPU Memory
/// Hierarchy through Microbenchmarking: https://arxiv.org/abs/1509.02308
/// Capturing the Memory Topology of GPUs: https://hgpu.org/?p=27501

/// From 1804.06826 Citadel Paper (https://arxiv.org/abs/1804.06826)
__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, double *dsink,
                      uint64_t *posArray) { // thread index


  uint32_t tid = threadIdx.x;

  // a register to avoid compiler optimization
  double sink = 0;

  // populate l1 cache to warm up
  #pragma unroll 1
  for (uint32_t i = tid; i < L1_SIZE; i += THREADS_NUM) {
    double *ptr = (double*)(posArray + i);
    asm volatile("{\n\t\t"
                 ".reg .f64 data;\n\t\t"
                 "ld.global.ca.f64 data, [%1];\n\t\t"
                 "add.f64 %0, data, %0;\n\t"
                 "}\n"
                 : "+d"(sink)
                 : "l"(ptr)
                 : "memory");
  }
  // synchronize all threads
  asm volatile("bar.sync 0;");

  // start timing
  uint32_t start = 0;
  asm volatile("mov.u32 %0, %%clock;" : "=r"(start)::"memory");

  // load data from l1 cache and accumulate
  #pragma unroll 1
  for (uint32_t i = 0; i < L1_SIZE; i += THREADS_NUM) {

    double *ptr = (double*)(posArray + i);

    // every warp loads all data in l1 cache

    for (uint32_t j = 0; j < THREADS_NUM; j += WARP_SIZE) {
      uint32_t offset = (tid + j) % THREADS_NUM;
      asm volatile("{\n\t\t"
                   ".reg .f64 data;\n\t\t"
                   "ld.global.ca.f64 data, [%1];\n\t\t"
                   "add.f64 %0, data, %0;\n\t"
                   "}\n"
                   : "+d"(sink)
                   : "l"(ptr + offset)
                   : "memory");
    }
  }
  // synchronize all threads
  asm volatile("bar.sync 0;");

  // stop timing
  uint32_t stop = 0;
  asm volatile("mov.u32 %0, %%clock;" : "=r"(stop)::"memory");

  // write time and data back to memory
  startClk[tid] = start;
  stopClk[tid] = stop;
  dsink[tid] = sink;
}
